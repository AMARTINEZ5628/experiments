
#include <hip/hip_runtime.h>
const double LATTICE_SPEED = 0.1;
const double TAU = 0.9;
const int DIRECTIONS = 9;
const int DIMENSIONS = 2;

#define KERNEL_HEADER(xvar, yvar, wvar, hvar) \
  const int x = blockIdx.x;\
  const int y = blockIdx.y;\
  const int width = gridDim.x;\
  const int height = gridDim.y
    

__global__ void stream(double *out, double *in) {
  KERNEL_HEADER(x, y, width, height);

  int z = 0;
  for(int i = -1; i <= 1; i++) {
    for(int j = -1; j <= 1; j++) {
        const int target = z + y * DIRECTIONS + x * DIRECTIONS * height;

        // Compute source index.
        int xs = x + i;
        int ys = y + j;
        const int source = z + ys * DIRECTIONS + xs * DIRECTIONS * height;
        if(xs >= 0 && xs < width && ys >= 0 && ys < height) {
            out[target] = in[source];
        } else {
            // If the check yielded out of bounds, that means that the 
            // node we're computing for is on an edge. Thus, we should 
            // use bounce-back -- instead of getting the value from a nearby node,
            // we get the value from itself.
            const int bounce_z = (1 - i) * 3 + (1 - j);
            const int bounce_src = bounce_z + y * DIRECTIONS + x * DIRECTIONS * height;
            const double friction_loss = 0.9;
            out[target] = friction_loss * in[bounce_src];
        }

        z++; 
    }
  }
}

__global__ void density(double *out, double* in) {
  KERNEL_HEADER(x, y, width, height);

  // Compute target index.
  const int target = y + x * height;
  out[target] = 0;

  int z = 0;
  for(int i = -1; i <= 1; i++) {
    for(int j = -1; j <= 1; j++) {
      // Compute source index.
      const int source = z + y * DIRECTIONS + x * DIRECTIONS * height;
      out[target] += in[source];
      z++;
    }
  }
}

__global__ void velocity(double *out, double* density, double* directions) {
  KERNEL_HEADER(x, y, width, height);

  // Compute target indices.
  const int target_x = 0 + y * DIMENSIONS + x * height * DIMENSIONS;
  const int target_y = 1 + y * DIMENSIONS + x * height * DIMENSIONS;
  out[target_x] = 0;
  out[target_y] = 0;

  int z = 0;
  for(int i = -1; i <= 1; i++) {
    for(int j = -1; j <= 1; j++) {
      const int source = z + y * DIRECTIONS + x * DIRECTIONS * height;
      out[target_x] += directions[source] * i;
      out[target_y] += directions[source] * j;
      z++;
    }
  }

  const int target = y + x * height;
  out[target_x] *= LATTICE_SPEED / density[target];
  out[target_y] *= LATTICE_SPEED / density[target];
}

__global__ void equilibrium(double *eq, double* density, double* velocity) {
  KERNEL_HEADER(x, y, width, height);

  const int density_src = y + x * height;
  const int xvel_src = 0 + y * DIMENSIONS + x * height * DIMENSIONS;
  const int yvel_src = 1 + y * DIMENSIONS + x * height * DIMENSIONS;

  // u . u
  double velmag = velocity[xvel_src] * velocity[xvel_src] + velocity[yvel_src] * velocity[yvel_src];

  int z = 0;
  for(int i = -1; i <= 1; i++) {
    for(int j = -1; j <= 1; j++) {
        // Compute the weight.
        double weight;
        if(i == 0 && j == 0) {
            weight = 4.0 / 9.0;
        } else if(i == 0 || j == 0) {
            weight = 1.0 / 9.0;
        } else {
            weight = 1.0 / 36.0;
        }

        // e_i . u
        double dotprod = i * velocity[xvel_src] + j * velocity[yvel_src];

        double sum = 1.0;
        sum += 3 / LATTICE_SPEED * dotprod;
        sum += 4.5 / (LATTICE_SPEED * LATTICE_SPEED) * dotprod * dotprod;
        sum -= 1.5 / (LATTICE_SPEED * LATTICE_SPEED) * velmag;

        const int target = z + y * DIRECTIONS + x * DIRECTIONS * height;
        eq[target] = weight * density[density_src] * sum;

        z++;
    }
  }
}

__global__ void update(double *out, double* equilibrium, double* directions) {
  KERNEL_HEADER(x, y, width, height);

  int z = 0;
  for(int i = -1; i <= 1; i++) {
    for(int j = -1; j <= 1; j++) {
        const int target = z + y * DIRECTIONS + x * DIRECTIONS * height;
        out[target] = directions[target] - (directions[target] - equilibrium[target]) / TAU;
        z++;
    }
  }
}
